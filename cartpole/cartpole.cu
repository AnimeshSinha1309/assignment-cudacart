#include <string>
#include <iostream>

#include <hip/hip_runtime.h>
#include <npp.h>

#include "image.hu"
#include "gpu.hu"
#include "generate.hu"

using namespace CudaProj::Cartpole;

int main(int argc, char** argv)
{
    auto device = initialize_gpu();
    auto works = print_cuda_info();
    std::string filename = "output.png";

    ImageMatrix cartpole(1000, ImageRow(1000));
    for (int i = 0; i < 1000; i++)
        for (int j = 0; j < 1000; j++)
            draw_scene(cartpole, i, j);

    CpuImage o_image(cartpole);
    CudaImage d_image(o_image);

    dim3 blocks_per_grid {10, 10, 1};
    dim3 threads_per_block{1, 1, 3};
    color_background<<<blocks_per_grid, threads_per_block>>>(
        d_image.image_ptr(), d_image.image_size().first, d_image.image_size().second);

    CpuImage h_image(d_image);
    auto matrix = o_image.to_matrix();
    save_image(matrix);
}
