#include "image.hu"

#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>

namespace CudaProj::Cartpole
{

// CPU image functions

CpuImage::CpuImage(int rows, int cols)
    : mRows(rows), mCols(cols)
{
    mImage = (Npp8u*) malloc(sizeof(Npp8u) * mCols * mRows * 3);
    mPitch = mCols * sizeof(Npp8u) * 3;
}

CpuImage::CpuImage(ImageMatrix image) : CpuImage(image.size(), image[0].size())
{
    for (int i = 0; i < mRows; i++)
    {
        for (int j = 0; j < mCols; j++)
        {
            mImage[(i * mCols + j) * 3 + 0] = image[i][j][0];
            mImage[(i * mCols + j) * 3 + 1] = image[i][j][1];
            mImage[(i * mCols + j) * 3 + 2] = image[i][j][2];
        }
    }
}

CpuImage::CpuImage(CudaImage image) : CpuImage(image.mRows, image.mCols)
{
    hipError_t eResult = hipMemcpy2D(
        mImage, mPitch, image.mImage, image.mPitch,
        mRows * sizeof(Npp8u), mCols, hipMemcpyDeviceToHost);
    if (eResult != hipSuccess)
        throw std::runtime_error("Cuda Memcpy from device to host failed." + eResult);

    int pixel1 = mImage[0], pixel2 = mImage[1], pixel3 = mImage[2];
    std::cout << "Data: " << pixel1 << ' ' << pixel2 << ' ' << pixel3 << std::endl;
}

CpuImage::~CpuImage()
{
    // free(mImage);
}

ImageMatrix CpuImage::to_matrix()
{
    auto matrix = ImageMatrix(mRows, std::vector<std::array<Npp8u, 3>>(mCols));
    for (int i = 0; i < mRows; i++)
    {
        for (int j = 0; j < mCols; j++)
        {
            matrix[i][j][0] = mImage[(i * mCols + j) * 3 + 0];
            matrix[i][j][1] = mImage[(i * mCols + j) * 3 + 1];
            matrix[i][j][2] = mImage[(i * mCols + j) * 3 + 2];
        }
    }
    return matrix;
}

// CUDA image functions

CudaImage::CudaImage(int width, int height)
    : mRows(width), mCols(height)
{
    mImage = nppiMalloc_8u_C3(mRows, mCols, &mPitch);
}

CudaImage::CudaImage(CpuImage image) : CudaImage(image.mRows, image.mCols)
{
    hipError_t eResult = hipMemcpy2D(
        mImage, mPitch, image.mImage, image.mPitch,
        mRows * sizeof(Npp8u), mCols, hipMemcpyHostToDevice);
    if (eResult != hipSuccess)
        throw std::runtime_error("Cuda Memcpy from host to device failed.");
}

CudaImage::~CudaImage()
{
    nppiFree(mImage);
}

Npp8u* CudaImage::image_ptr()
{
    return mImage;
}

std::pair<int, int> CudaImage::image_size()
{
    return {mRows, mCols};
}

void save_image(ImageMatrix image)
{
    cv::Mat output(image.size(), image[0].size(), CV_8UC3);
    for (int row = 0; row < output.rows; ++row) {
        for (int col = 0; col < output.cols; ++col) {
            cv::Vec3b& pixel = output.at<cv::Vec3b>(row, col);
            pixel[0] = image[row][col][0];
            pixel[1] = image[row][col][1];
            pixel[2] = image[row][col][2];
        }
    }
    cv::imwrite("cartpole_image.png", output);
}

}
